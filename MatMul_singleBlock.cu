#include "hip/hip_runtime.h"
/**
This is an exmple code used in the CUDA Lecture 2 (Quick Lab. 9-2) <br>
@author : Duksu Kim
*/

#include "hip/hip_runtime.h"
#include ""

#include <chrono>
typedef std::chrono::high_resolution_clock Clock;
//#include <DS_timer.h>

#include <omp.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define NUM_CPU_THREADS	(4)

#define ROW_SIZE (32)
#define K_SIZE   (128)
#define COL_SIZE (32)

#if K_SIZE < 129
#define USE_SHARED_VER
#endif
//#define USE_SHARED_VER_C
#define USE_BASE_KERNEL

#define	WORK_LOAD (1024)
#define MAT_SIZE_A (ROW_SIZE*K_SIZE)
#define MAT_SIZE_B (K_SIZE*COL_SIZE)
#define MAT_SIZE_C (ROW_SIZE*COL_SIZE)

// input matrix
float A[ROW_SIZE][K_SIZE];	// m * k
float B[K_SIZE][COL_SIZE];	// k * n

// timer
//DS_timer* timer;
//void setTimer(void);
//#define TIMER_HOST		0
//#define TIMER_KERNEL	1
//#define TIMER_KERNEL_SH	2
//#define TIMER_KERNEL_SH_C	3
//#define TIMER_HtoD		4
//#define TIMER_DtoH		5
//#define NUM_TIMER		(TIMER_DtoH+1)

void genInputMatrices(void);

// output matrix
float hostC[ROW_SIZE][COL_SIZE];	// host result
float deviceC[ROW_SIZE][COL_SIZE];	// device result

#define memsetZero(_P,_type,_size) memset(_P, 0, sizeof(_type)*_size);
#define dMemAlloc(_P, _type, _size) hipMalloc(&_P, sizeof(_type)*_size);

#ifdef USE_BASE_KERNEL
__global__ void matMul_kernel(float* _A, float* _B, float* _C)
{
	int row = threadIdx.y;
	int col = threadIdx.x;
	int index = row * blockDim.x + col;

	_C[index] = 0;
	for (int k = 0; k < K_SIZE; k++)
		for (int i = 0; i < WORK_LOAD; i++)
			_C[index] += _A[row * K_SIZE + k] * _B[col + k * COL_SIZE];
}

__global__ void matMul_kernel_xRow(float* _A, float* _B, float* _C)
{
	int row = threadIdx.x;
	int col = threadIdx.y;
	int index = row * blockDim.y + col;

	_C[index] = 0;
	for (int k = 0; k < K_SIZE; k++)
		for (int i = 0; i < WORK_LOAD; i++)
			_C[index] += _A[row * K_SIZE + k] * _B[col + k * COL_SIZE];
}

#endif


void main(void)
{
	//timer = NULL;	setTimer();

	float *dA, *dB, *dC;
	dA = dB = dC = NULL;

	memsetZero(A, float, MAT_SIZE_A);	memsetZero(B, float, MAT_SIZE_B);
	memsetZero(hostC, float, MAT_SIZE_C);	memsetZero(deviceC, float, MAT_SIZE_C);

	// device memory allocaiton
	dMemAlloc(dA, float, MAT_SIZE_A);
	dMemAlloc(dB, float, MAT_SIZE_B);
	dMemAlloc(dC, float, MAT_SIZE_C);

	// generate input matrices
	genInputMatrices();

	// Host code
	//timer->onTimer(TIMER_HOST);
	//#pragma omp parallel for num_threads(NUM_CPU_THREADS)
	for (int r = 0; r < ROW_SIZE; r++)
		for (int c = 0; c < COL_SIZE; c++)
			for (int k = 0; k < K_SIZE; k++)
				for (int i = 0; i < WORK_LOAD; i++)
					hostC[r][c] += A[r][k] * B[k][c];
	//timer->offTimer(TIMER_HOST);

	// Copy input matrices : H -> D
	//timer->onTimer(TIMER_HtoD);
	hipMemcpy(dA, A, sizeof(float)*MAT_SIZE_A, hipMemcpyHostToDevice);
	hipMemcpy(dB, B, sizeof(float)*MAT_SIZE_B, hipMemcpyHostToDevice);
	//timer->offTimer(TIMER_HtoD);

	dim3 blockDim(COL_SIZE, ROW_SIZE);

#ifdef USE_BASE_KERNEL
	//// Kernel call
	//timer->onTimer(TIMER_KERNEL);

	matMul_kernel << <1, blockDim >> > (dA, dB, dC);
	hipDeviceSynchronize();

	// For matMul_kernel_xRow
	//blockDim.x = ROW_SIZE;
	//blockDim.y = COL_SIZE;
	//matMul_kernel_xRow <<<1, blockDim >> > (dA, dB, dC);
	//hipDeviceSynchronize();

	//timer->offTimer(TIMER_KERNEL);
#endif


	// Get back result : D -> H
	//timer->onTimer(TIMER_DtoH);
	hipMemcpy(deviceC, dC, sizeof(float)*MAT_SIZE_C, hipMemcpyDeviceToHost);
	//timer->onTimer(TIMER_DtoH);

	// check the results
	bool isCorrect = true;

	float *pHostC = &hostC[0][0];
	float *pDeviceC = &deviceC[0][0];

	for (int i = 0; i < MAT_SIZE_C; i++) {
		if (pHostC[i] != pDeviceC[i]) {
			printf("[%d] %.2f, %.2f\n", i, pHostC[i], pDeviceC[i]);
			isCorrect = false;
			break;
		}
	}

	if (isCorrect) printf("Result is correct!\n");
	else printf("Result is not correct!!!!!!\n");

	//timer->printTimer();
	//if (timer != NULL)
	//	delete timer;
}

void genInputMatrices(void)
{
	for (int r = 0; r < ROW_SIZE; r++)
		for (int k = 0; k < K_SIZE; k++)
			A[r][k] = rand() % 100;

	for (int k = 0; k < K_SIZE; k++)
		for (int c = 0; c < COL_SIZE; c++)
			B[k][c] = rand() % 100;

	// currently disable it for making a simple example while avoiding precision problem
	// A[r][c] += (float)(rand() % 100) / 100.0;
	// B[r][c] += (float)(rand() % 100) / 100.0;
}

//void setTimer(void)
//{
//	timer = new DS_timer(NUM_TIMER);
//
//	timer->initTimers();
//	timer->setTimerName(TIMER_HOST, "CPU code");
//	timer->setTimerName(TIMER_KERNEL, "Kernel launch");
//	timer->setTimerName(TIMER_KERNEL_SH, "Kernel launch (shared ver.)");
//	timer->setTimerName(TIMER_KERNEL_SH_C, "Kernel launch (shared ver. C)");
//	timer->setTimerName(TIMER_HtoD, "[Data transter] host->device");
//	timer->setTimerName(TIMER_DtoH, "[Data transfer] device->host");
//}
